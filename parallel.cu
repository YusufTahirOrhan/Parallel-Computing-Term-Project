#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include "parallel.h"
#include ""
#include <fstream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// CUDA kernel to calculate gradients
__global__ void calculateGradientCUDA(const uint16_t* d_image, uint8_t* d_gradient, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Boundary check
	if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
		int idx = y * width + x;
		uint16_t center = d_image[idx];

		// Read 8 neighbors
		uint16_t neighbors[8] = {
			d_image[idx - width],        // N
			d_image[idx - width + 1],    // NE
			d_image[idx + 1],            // E
			d_image[idx + width + 1],    // SE
			d_image[idx + width],        // S
			d_image[idx + width - 1],    // SW
			d_image[idx - 1],            // W
			d_image[idx - width - 1]     // NW
		};

		// According to the problem statement:
		// If the pixel is 'lighter' (with a smaller intensity value) than all neighbors, direction=0.
		// Otherwise, point to the lightest neighbor (the one with the smallest intensity).
		uint16_t minValue = center;
		int direction = 0;
		for (int i = 0; i < 8; ++i) {
			if (neighbors[i] < minValue) {
				minValue = neighbors[i];
				direction = i + 1; // (1..8)
			}
		}
		d_gradient[idx] = direction;
	}
}

// CUDA kernel to calculate updated labels in a 7x7 neighborhood
__global__ void calculate7x7NeighborhoodCUDA(const uint8_t* d_gradient, uint8_t* d_newLabels, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Boundary check (3 pixels margin for 7x7)
	if (x >= 3 && x < width - 3 && y >= 3 && y < height - 3) {
		float sumX = 0.0f;
		float sumY = 0.0f;

		// Accumulate direction vectors within the 7x7 neighborhood
		for (int ky = -3; ky <= 3; ++ky) {
			for (int kx = -3; kx <= 3; ++kx) {
				int neighborIdx = (y + ky) * width + (x + kx);
				int direction = d_gradient[neighborIdx];

				// Convert direction (1..8) into a unit vector
				switch (direction) {
				case 1: sumY -= 1.0f;          break; // N
				case 2: sumY -= 1.0f; sumX += 1.0f; break; // NE
				case 3: sumX += 1.0f;          break; // E
				case 4: sumY += 1.0f; sumX += 1.0f; break; // SE
				case 5: sumY += 1.0f;          break; // S
				case 6: sumY += 1.0f; sumX -= 1.0f; break; // SW
				case 7: sumX -= 1.0f;          break; // W
				case 8: sumY -= 1.0f; sumX -= 1.0f; break; // NW
				}
			}
		}

		// Calculate magnitude & angle
		float magnitude = sqrtf(sumX * sumX + sumY * sumY);
		int newDirection = 0;

		if (magnitude > 0.0f) {
			float angle = atan2f(sumY, sumX) * 180.0f / M_PI;
			// Determine direction based on angle
			if (angle >= -22.5f && angle < 22.5f)    newDirection = 3; // E
			else if (angle >= 22.5f && angle < 67.5f)    newDirection = 2; // NE
			else if (angle >= 67.5f && angle < 112.5f)   newDirection = 1; // N
			else if (angle >= 112.5f && angle < 157.5f)   newDirection = 8; // NW
			else if (angle >= -67.5f && angle < -22.5f)   newDirection = 4; // SE
			else if (angle >= -112.5f && angle < -67.5f)   newDirection = 5; // S
			else if (angle >= -157.5f && angle < -112.5f)  newDirection = 6; // SW
			else                                          newDirection = 7; // W
		}

		int idx = y * width + x;
		d_newLabels[idx] = newDirection;
	}
}

// ---------------------------------------------------------------------------------
// CUDA kernel to find endpoints for each pixel. Instead of storing
// the entire path, we only store (startX, startY, endX, endY).
// 
// Added a 'step limit' to avoid infinite loops if there's a cycle.
// ---------------------------------------------------------------------------------
__global__ void findEndpointsCUDA(const uint8_t* d_directions, int* d_endpoints, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;

	int idx = y * width + x;

	// Store the starting pixel (beginning of the path)
	int startX = x;
	int startY = y;

	// We will follow the gradient directions until we reach:
	// - direction=0 (local extremum)
	// - boundary
	// - or step limit (to break a potential cycle)
	int currentX = x;
	int currentY = y;

	// Step limit to avoid infinite loops in case of a cycle
	int steps = 0;
	const int maxSteps = width * height; // or a smaller heuristic, e.g. 2*(width+height)

	while (true) {
		// If steps exceed this limit, break to avoid infinite loops
		if (++steps > maxSteps) {
			break;
		}

		int direction = d_directions[currentY * width + currentX];
		if (direction == 0) {
			// local minimum or no smaller neighbor
			break;
		}

		int nextX = currentX;
		int nextY = currentY;
		switch (direction) {
		case 1:  nextY--;            break; // N
		case 2:  nextY--; nextX++;   break; // NE
		case 3:  nextX++;            break; // E
		case 4:  nextY++; nextX++;   break; // SE
		case 5:  nextY++;            break; // S
		case 6:  nextY++; nextX--;   break; // SW
		case 7:  nextX--;            break; // W
		case 8:  nextY--; nextX--;   break; // NW
		default:
			// Unexpected direction; break out.
			break;
		}

		// If we go out of bounds, stop
		if (nextX < 0 || nextX >= width || nextY < 0 || nextY >= height) {
			break;
		}

		// Move forward
		currentX = nextX;
		currentY = nextY;
	}

	// Final endpoint
	int endX = currentX;
	int endY = currentY;

	// Write to global memory: each pixel has 4 integers
	// (startX, startY, endX, endY)
	d_endpoints[4 * idx + 0] = startX;
	d_endpoints[4 * idx + 1] = startY;
	d_endpoints[4 * idx + 2] = endX;
	d_endpoints[4 * idx + 3] = endY;
}

 // ---------------------------------------------------------------------------------
 // processTileParallel:
 // Processes one tile using the above kernels.
 // 
 // 1) Gradient calculation
 // 2) 7x7 neighborhood average direction
 // 3) Endpoint detection with step limit
 // ---------------------------------------------------------------------------------
void processTileParallel(const cv::Mat& tile, cv::Mat& newLabels, std::vector<std::vector<std::pair<int, int>>>& paths)
{
	int width = tile.cols;
	int height = tile.rows;

	// Memory sizes
	size_t imageSize = width * height * sizeof(uint16_t);
	size_t gradientSize = width * height * sizeof(uint8_t);
	size_t endpointsSize = width * height * 4 * sizeof(int);

	// Allocate device memory
	uint16_t* d_image = nullptr;
	uint8_t* d_gradient = nullptr;
	uint8_t* d_newLabels = nullptr;
	int* d_endpoints = nullptr;

	hipMalloc(&d_image, imageSize);
	hipMalloc(&d_gradient, gradientSize);
	hipMalloc(&d_newLabels, gradientSize);
	hipMalloc(&d_endpoints, endpointsSize);

	// Copy tile data to device
	hipMemcpy(d_image, tile.data, imageSize, hipMemcpyHostToDevice);
	hipMemset(d_endpoints, -1, endpointsSize);

	// You can adjust block size according to your GPU
	dim3 blockSize(16, 16);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
		(height + blockSize.y - 1) / blockSize.y);

	// 1) Gradient kernel
	calculateGradientCUDA << <gridSize, blockSize >> > (d_image, d_gradient, width, height);
	hipDeviceSynchronize();

	// 2) 7x7 neighborhood kernel
	calculate7x7NeighborhoodCUDA << <gridSize, blockSize >> > (d_gradient, d_newLabels, width, height);
	hipDeviceSynchronize();

	// 3) Find endpoints
	findEndpointsCUDA << <gridSize, blockSize >> > (d_newLabels, d_endpoints, width, height);
	hipDeviceSynchronize();

	// Copy final labels (for visualization or debugging)
	newLabels = cv::Mat(tile.size(), CV_8U);
	hipMemcpy(newLabels.data, d_newLabels, gradientSize, hipMemcpyDeviceToHost);

	// Copy endpoint data
	std::vector<int> h_endpoints(width * height * 4, -1);
	hipMemcpy(h_endpoints.data(), d_endpoints, endpointsSize, hipMemcpyDeviceToHost);

	// Convert endpoint data to a vector of vectors of pairs
	// each pixel => [(startX, startY), (endX, endY)]
	paths.clear();
	paths.reserve(width * height);

	for (int i = 0; i < width * height; ++i) {
		int startX = h_endpoints[4 * i + 0];
		int startY = h_endpoints[4 * i + 1];
		int endX = h_endpoints[4 * i + 2];
		int endY = h_endpoints[4 * i + 3];

		if (startX >= 0 && startY >= 0) {
			std::vector<std::pair<int, int>> path(2);
			path[0] = { startX, startY };
			path[1] = { endX,   endY };
			paths.push_back(path);
		}
	}

	// Free device memory
	hipFree(d_image);
	hipFree(d_gradient);
	hipFree(d_newLabels);
	hipFree(d_endpoints);
}

 // ---------------------------------------------------------------------------------
 // processImageWithTiling:
 // Splits the large image into tiles, processes each tile on the GPU,
 // and combines / logs results.
 // ---------------------------------------------------------------------------------
void processImageWithTiling(const cv::Mat& image, const std::string& outputPath, int tileSize, int overlap)
{
	int width = image.cols;
	int height = image.rows;

	// Prepare an output image (just to store new directions if needed)
	cv::Mat combinedNewLabels = cv::Mat::zeros(image.size(), CV_8U);

	// For storing all paths from all tiles
	std::vector<std::vector<std::pair<int, int>>> allPaths;
	allPaths.reserve(width * height);

	// Iterate over tiles
	for (int y = 0; y < height; y += tileSize) {
		for (int x = 0; x < width; x += tileSize) {
			// Calculate tile boundaries including overlap
			int startX = std::max(0, x - overlap);
			int startY = std::max(0, y - overlap);
			int endX = std::min(width, x + tileSize + overlap);
			int endY = std::min(height, y + tileSize + overlap);

			// Extract the tile region from the original image
			cv::Rect tileRegion(startX, startY, endX - startX, endY - startY);
			cv::Mat extendedTile = image(tileRegion);

			// Process the tile on GPU
			cv::Mat tileNewLabels;
			std::vector<std::vector<std::pair<int, int>>> tilePaths;
			processTileParallel(extendedTile, tileNewLabels, tilePaths);

			// Copy tile result back to the combined image (excluding overlap)
			cv::Rect processingRegion(
				x - startX,
				y - startY,
				std::min(tileSize, width - x),
				std::min(tileSize, height - y)
			);
			cv::Mat tileResultROI = tileNewLabels(processingRegion);
			tileResultROI.copyTo(combinedNewLabels(cv::Rect(x, y, tileResultROI.cols, tileResultROI.rows)));

			// Adjust paths to global coordinates (because tilePaths are local)
			for (auto& path : tilePaths) {
				for (auto& point : path) {
					point.first += startX; // offset X
					point.second += startY; // offset Y
				}
			}
			allPaths.insert(allPaths.end(), tilePaths.begin(), tilePaths.end());
		}
	}

	// Save (start,end) pairs to a text file
	std::ofstream outputFile(outputPath + ".txt");
	if (outputFile.is_open()) {
		for (const auto& p : allPaths) {
			if (p.size() == 2) {
				outputFile << "(" << p[0].first << "," << p[0].second << ") "
					<< "(" << p[1].first << "," << p[1].second << ")\n";
			}
		}
		outputFile.close();
		std::cout << "Gradient paths saved as '" << outputPath << ".txt'." << std::endl;
	}
	else {
		std::cerr << "Could not open file for writing: " << outputPath << ".txt" << std::endl;
	}
}
